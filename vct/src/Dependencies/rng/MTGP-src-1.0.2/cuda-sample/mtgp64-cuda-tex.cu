#include "hip/hip_runtime.h"
/*
 * Sample Program for CUDA 2.3
 * written by M.Saito (saito@math.sci.hiroshima-u.ac.jp)
 *
 * This sample uses texture reference.
 * The generation speed of PRNG using texture is faster than using
 * constant tabel on Geforce GTX 260.
 *
 * MTGP64-11213
 * This program generates 64-bit unsigned integers.
 * The period of generated integers is 2<sup>11213</sup>-1.
 * This also generates double precision floating point numbers.
 */
#define __STDC_FORMAT_MACROS 1
#define __STDC_CONSTANT_MACROS 1
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <stdlib.h>
extern "C" {
#include "mtgp64-fast.h"
#include "mtgp64dc-param-11213.c"
}
#define MEXP 11213
#define N MTGPDC_N
#define THREAD_NUM MTGPDC_FLOOR_2P
#define LARGE_SIZE (THREAD_NUM * 3)
#define PARAM_NUM_MAX mtgpdc_params_11213_num
#define BLOCK_NUM_MAX 200
#define TBL_SIZE 16

/**
 * kernel I/O
 * This structure must be initialized before first use.
 */
struct mtgp64_kernel_status_t {
    uint64_t status[N];
};

/*
 * Texture References.
 */
texture<uint32_t, 1, hipReadModeElementType> tex_param_ref;
texture<uint32_t, 1, hipReadModeElementType> tex_temper_ref;
texture<uint32_t, 1, hipReadModeElementType> tex_double_ref;

/*
 * Generator Parameters.
 */
__constant__ uint32_t pos_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t sh1_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t sh2_tbl[BLOCK_NUM_MAX];
/* high_mask and low_mask should be set by make_constant(), but
 * did not work.
 */
__constant__ uint32_t high_mask = 0xfff80000;
__constant__ uint32_t low_mask =  0x00000000;

/**
 * Shared memory
 * The generator's internal status vector.
 */
__shared__ uint32_t status[2][LARGE_SIZE]; /* 512 * 3 elements, 12288 bytes. */

/**
 * The function of the recursion formula calculation.
 *
 * @param RH 32-bit MSBs of output
 * @param RL 32-bit LSBs of output
 * @param X1H MSBs of the farthest part of state array.
 * @param X1L LSBs of the farthest part of state array.
 * @param X2H MSBs of the second farthest part of state array.
 * @param X2L LSBs of the second farthest part of state array.
 * @param YH MSBs of a part of state array.
 * @param YL LSBs of a part of state array.
 * @param bid block id.
 */
__device__ void para_rec(uint32_t *RH,
			 uint32_t *RL,
			 uint32_t X1H,
			 uint32_t X1L,
			 uint32_t X2H,
			 uint32_t X2L,
			 uint32_t YH,
			 uint32_t YL,
			 int bid) {
    uint32_t XH = (X1H & high_mask) ^ X2H;
    uint32_t XL = (X1L & low_mask) ^ X2L;
    uint32_t MAT;

    XH ^= XH << sh1_tbl[bid];
    XL ^= XL << sh1_tbl[bid];
    YH = XL ^ (YH >> sh2_tbl[bid]);
    YL = XH ^ (YL >> sh2_tbl[bid]);
    MAT = tex1Dfetch(tex_param_ref, bid * 16 + (YL & 0x0f));
    *RH = YH ^ MAT;
    *RL = YL;
}

/**
 * The tempering function.
 *
 * @param VH MSBs of the output value should be tempered.
 * @param VL LSBs of the output value should be tempered.
 * @param TL LSBs of the tempering helper value.
 * @param bid block id.
 * @return the tempered value.
 */
__device__ uint64_t temper(uint32_t VH,
			   uint32_t VL,
			   uint32_t TL,
			   int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = tex1Dfetch(tex_temper_ref, bid * 16 + (TL & 0x0f));
    VH ^= MAT;
    r = ((uint64_t)VH << 32) | VL;
    return r;
}

/**
 * The tempering and converting function.
 * By using the presetted table, converting to IEEE format
 * and tempering are done simultaneously.
 *
 * @param VH MSBs of the output value should be tempered.
 * @param VL LSBs of the output value should be tempered.
 * @param TL LSBs of the tempering helper value.
 * @param bid block id.
 * @return the tempered and converted value.
 */
__device__ uint64_t temper_double(uint32_t VH,
				  uint32_t VL,
				  uint32_t TL,
				  int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = tex1Dfetch(tex_double_ref, bid * 16 + (TL & 0x0f));
    r = ((uint64_t)VH << 32) | VL;
    r = (r >> 12) ^ ((uint64_t)MAT << 32);
    return r;
}

/**
 * Read the internal state vector from kernel I/O data, and
 * put them into shared memory.
 *
 * @param status shared memory.
 * @param d_status kernel I/O data
 * @param bid block id
 * @param tid thread id
 */
__device__ void status_read(uint32_t status[2][LARGE_SIZE],
			    const mtgp64_kernel_status_t *d_status,
			    int bid,
			    int tid) {
    uint64_t x;

    x = d_status[bid].status[tid];
    status[0][LARGE_SIZE - N + tid] = x >> 32;
    status[1][LARGE_SIZE - N + tid] = x & 0xffffffff;
    if (tid < N - THREAD_NUM) {
	x = d_status[bid].status[THREAD_NUM + tid];
	status[0][LARGE_SIZE - N + THREAD_NUM + tid] = x >> 32;
	status[1][LARGE_SIZE - N + THREAD_NUM + tid] = x & 0xffffffff;
    }
    __syncthreads();
}

/**
 * Read the internal state vector from shared memory, and
 * write them into kernel I/O data.
 *
 * @param status shared memory.
 * @param d_status kernel I/O data
 * @param bid block id
 * @param tid thread id
 */
__device__ void status_write(mtgp64_kernel_status_t *d_status,
			     const uint32_t status[2][LARGE_SIZE],
			     int bid,
			     int tid) {
    uint64_t x;

    x = (uint64_t)status[0][LARGE_SIZE - N + tid] << 32;
    x = x | status[1][LARGE_SIZE - N + tid];
    d_status[bid].status[tid] = x;
    if (tid < N - THREAD_NUM) {
	x = (uint64_t)status[0][4 * THREAD_NUM - N + tid] << 32;
	x = x | status[1][4 * THREAD_NUM - N + tid];
	d_status[bid].status[THREAD_NUM + tid] = x;
    }
    __syncthreads();
}

/**
 * kernel function.
 * This function generates 64-bit unsigned integers in d_data
 *
 * @params d_status kernel I/O data
 * @params d_data output
 * @params size number of output data requested.
 */
__global__ void mtgp64_uint64_kernel(mtgp64_kernel_status_t* d_status,
				     uint64_t* d_data, int size) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t YH;
    uint32_t YL;
    uint64_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {

#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[0][LARGE_SIZE - N + tid]:%08x\n",
		   status[0][LARGE_SIZE - N + tid]);
	    printf("status[1][LARGE_SIZE - N + tid]:%08x\n",
		   status[1][LARGE_SIZE - N + tid]);
	    printf("status[0][LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[0][LARGE_SIZE - N + tid + 1]);
	    printf("status[1][LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[1][LARGE_SIZE - N + tid + 1]);
	    printf("status[0][LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[0][LARGE_SIZE - N + tid + pos]);
	    printf("status[1][LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[1][LARGE_SIZE - N + tid + pos]);
	    printf("sh1:%d\n", sh1_tbl[bid]);
	    printf("sh2:%d\n", sh2_tbl[bid]);
	    printf("high_mask:%08x\n", high_mask);
	    printf("low_mask:%08x\n", low_mask);
	    for (int j = 0; j < 16; j++) {
		printf("tbl[%d]:%08x\n", j, param_tbl[0][j]);
	    }
	}
#endif
	para_rec(&YH,
		 &YL,
		 status[0][LARGE_SIZE - N + tid],
		 status[1][LARGE_SIZE - N + tid],
		 status[0][LARGE_SIZE - N + tid + 1],
		 status[1][LARGE_SIZE - N + tid + 1],
		 status[0][LARGE_SIZE - N + tid + pos],
		 status[1][LARGE_SIZE - N + tid + pos],
		 bid);
	status[0][tid] = YH;
	status[1][tid] = YL;
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[0][tid]:%08x\n",	status[0][tid]);
	    printf("status[1][tid]:%08x\n",	status[1][tid]);
	}
#endif
	o = temper(YH,
		   YL,
		   status[1][LARGE_SIZE - N + tid + pos - 1],
		   bid);
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("o:%016" PRIx64 "\n", o);
	}
#endif
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + THREAD_NUM] = YH;
	status[1][tid + THREAD_NUM] = YL;
	o = temper(YH,
		   YL,
		   status[1][(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
		   bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][2 * THREAD_NUM - N + tid],
		 status[1][2 * THREAD_NUM - N + tid],
		 status[0][2 * THREAD_NUM - N + tid + 1],
		 status[1][2 * THREAD_NUM - N + tid + 1],
		 status[0][2 * THREAD_NUM - N + tid + pos],
		 status[1][2 * THREAD_NUM - N + tid + pos],
		 bid);
	status[0][tid + 2 * THREAD_NUM] = YH;
	status[1][tid + 2 * THREAD_NUM] = YL;
	o = temper(YH,
		   YL,
		   status[1][tid + pos - 1 + 2 * THREAD_NUM - N],
		   bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * kernel function.
 * This function generates double precision floating point numbers in d_data.
 *
 * @params d_status kernel I/O data
 * @params d_data output. IEEE double precision format.
 * @params size number of output data requested.
 */
__global__ void mtgp64_double_kernel(mtgp64_kernel_status_t* d_status,
				     uint64_t* d_data, int size)
{

    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t YH;
    uint32_t YL;
    uint64_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {
	para_rec(&YH,
		 &YL,
		 status[0][LARGE_SIZE - N + tid],
		 status[1][LARGE_SIZE - N + tid],
		 status[0][LARGE_SIZE - N + tid + 1],
		 status[1][LARGE_SIZE - N + tid + 1],
		 status[0][LARGE_SIZE - N + tid + pos],
		 status[1][LARGE_SIZE - N + tid + pos],
		 bid);
	status[0][tid] = YH;
	status[1][tid] = YL;
	o = temper_double(YH,
			  YL,
			  status[1][LARGE_SIZE - N + tid + pos - 1],
			  bid);
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + THREAD_NUM] = YH;
	status[1][tid + THREAD_NUM] = YL;
	o = temper_double(
	    YH,
	    YL,
	    status[1][(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
	    bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][2 * THREAD_NUM - N + tid],
		 status[1][2 * THREAD_NUM - N + tid],
		 status[0][2 * THREAD_NUM - N + tid + 1],
		 status[1][2 * THREAD_NUM - N + tid + 1],
		 status[0][2 * THREAD_NUM - N + tid + pos],
		 status[1][2 * THREAD_NUM - N + tid + pos],
		 bid);
	status[0][tid + 2 * THREAD_NUM] = YH;
	status[1][tid + 2 * THREAD_NUM] = YL;
	o = temper_double(YH,
			  YL,
			  status[1][tid + pos - 1 + 2 * THREAD_NUM - N],
			  bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * This function sets constants in device memory.
 * @param params input, MTGP64 parameters.
 */
void make_constant(const mtgp64_params_fast_t params[],
		   int block_num) {
    const int size1 = sizeof(uint32_t) * block_num;
    uint32_t *h_pos_tbl;
    uint32_t *h_sh1_tbl;
    uint32_t *h_sh2_tbl;
#if 0
    uint32_t *h_high_mask;
    uint32_t *h_low_mask;
#endif
    h_pos_tbl = (uint32_t *)malloc(size1);
    h_sh1_tbl = (uint32_t *)malloc(size1);
    h_sh2_tbl = (uint32_t *)malloc(size1);
#if 0
    h_high_mask = (uint32_t *)malloc(sizeof(uint32_t));
    h_low_mask = (uint32_t *)malloc(sizeof(uint32_t));
#endif
    if (h_pos_tbl == NULL
	|| h_sh1_tbl == NULL
	|| h_sh2_tbl == NULL
#if 0
	|| h_high_mask == NULL
	|| h_low_mask == NULL
#endif
	) {
	printf("failure in allocating host memory for constant table.\n");
	exit(1);
    }
#if 0
    *h_high_mask = params[0].mask >> 32;
    *h_low_mask = params[0].mask & 0xffffffffU;
#endif
    for (int i = 0; i < block_num; i++) {
	h_pos_tbl[i] = params[i].pos;
	h_sh1_tbl[i] = params[i].sh1;
	h_sh2_tbl[i] = params[i].sh2;
    }
    // copy from malloc area only
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pos_tbl), h_pos_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sh1_tbl), h_sh1_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sh2_tbl), h_sh2_tbl, size1));
#if 0
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&high_mask),
				      &h_high_mask, sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&low_mask),
				      &h_low_mask, sizeof(uint32_t)));
#endif
    free(h_pos_tbl);
    free(h_sh1_tbl);
    free(h_sh2_tbl);
#if 0
    free(h_high_mask);
    free(h_low_mask);
#endif
}

/**
 * This function sets constants in device memory.
 * @param params input, MTGP64 parameters.
 */
void make_texture(const mtgp64_params_fast_t params[],
		  uint32_t *d_texture_tbl[3],
		  int block_num) {
    const int count = block_num * TBL_SIZE;
    const int size = sizeof(uint32_t) * count;
    uint32_t *h_texture_tbl[3];
    int i, j;
    for (i = 0; i < 3; i++) {
	h_texture_tbl[i] = (uint32_t *)malloc(size);
	if (h_texture_tbl[i] == NULL) {
	    for (j = 0; j < i; j++) {
		free(h_texture_tbl[i]);
	    }
	    printf("failure in allocating host memory for constant table.\n");
	    exit(1);
	}
    }
    for (int i = 0; i < block_num; i++) {
	for (int j = 0; j < TBL_SIZE; j++) {
	    h_texture_tbl[0][i * TBL_SIZE + j] = params[i].tbl[j] >> 32;
	    h_texture_tbl[1][i * TBL_SIZE + j] = params[i].tmp_tbl[j] >> 32;
	    h_texture_tbl[2][i * TBL_SIZE + j] = params[i].dbl_tmp_tbl[j] >> 32;
	}
    }
    CUDA_SAFE_CALL(hipMemcpy(d_texture_tbl[0], h_texture_tbl[0], size,
			      hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_texture_tbl[1], h_texture_tbl[1], size,
			      hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_texture_tbl[2], h_texture_tbl[2], size,
			      hipMemcpyHostToDevice));
    tex_param_ref.filterMode = hipFilterModePoint;
    tex_temper_ref.filterMode = hipFilterModePoint;
    tex_double_ref.filterMode = hipFilterModePoint;
    CUDA_SAFE_CALL(hipBindTexture(0, tex_param_ref, d_texture_tbl[0], size));
    CUDA_SAFE_CALL(hipBindTexture(0, tex_temper_ref, d_texture_tbl[1], size));
    CUDA_SAFE_CALL(hipBindTexture(0, tex_double_ref, d_texture_tbl[2], size));
    free(h_texture_tbl[0]);
    free(h_texture_tbl[1]);
    free(h_texture_tbl[2]);
}

#include "mtgp-cuda-common.c"
#include "mtgp64-cuda-common.c"

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param d_status kernel I/O data.
 * @param num_data number of data to be generated.
 */
void make_uint64_random(mtgp64_kernel_status_t* d_status,
			int num_data,
			int block_num) {
    uint64_t* d_data;
    unsigned int timer = 0;
    uint64_t* h_data;
    hipError_t e;
    float gputime;

    printf("generating 64-bit unsigned random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(uint64_t) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (uint64_t *) malloc(sizeof(uint64_t) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp64_uint64_kernel<<< block_num, THREAD_NUM>>>(
	d_status, d_data, num_data / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint64_t) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_uint64_array(h_data, num_data, block_num);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param d_status kernel I/O data.
 * @param num_data number of data to be generated.
 */
void make_double_random(mtgp64_kernel_status_t* d_status,
			int num_data,
			int block_num) {
    uint64_t* d_data;
    unsigned int timer = 0;
    double* h_data;
    hipError_t e;
    float gputime;

    printf("generating double precision floating point random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(uint64_t) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (double *) malloc(sizeof(double) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp64_double_kernel<<< block_num, THREAD_NUM >>>(
	d_status, d_data, num_data / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint64_t) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_double_array(h_data, num_data, block_num);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

int main(int argc, char** argv)
{
    // LARGE_SIZE is a multiple of 16
    int num_data = 10000000;
    int block_num;
    int block_num_max;
    int num_unit;
    int r;
    mtgp64_kernel_status_t* d_status;
    uint32_t *d_texture[3];

    if (argc >= 2) {
	errno = 0;
	block_num = strtol(argv[1], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	if (BLOCK_NUM_MAX < PARAM_NUM_MAX) {
	    block_num_max = BLOCK_NUM_MAX;
	} else {
	    block_num_max = PARAM_NUM_MAX;
	}
	if (block_num < 1 || block_num > block_num_max) {
	    printf("%s block_num should be between 1 and %d\n",
		   argv[0], block_num_max);
	    return 1;
	}
	errno = 0;
	num_data = strtol(argv[2], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	argc -= 2;
	argv += 2;
    } else {
	CUT_DEVICE_INIT(argc, argv);
	printf("%s number_of_block number_of_output\n", argv[0]);
	block_num = get_suitable_block_num(sizeof(uint64_t),
					   THREAD_NUM,
					   LARGE_SIZE);
	if (block_num <= 0) {
	    printf("can't calculate sutable number of blocks.\n");
	    return 1;
	}
	printf("the suitable number of blocks for device 0 "
	       "will be multiple of %d\n", block_num);
	return 1;
    }
    CUT_DEVICE_INIT(argc, argv);
    num_unit = LARGE_SIZE * block_num;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_status,
			      sizeof(mtgp64_kernel_status_t) * block_num));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_texture[0],
			      sizeof(uint32_t) * block_num * TBL_SIZE));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_texture[1],
			      sizeof(uint32_t) * block_num * TBL_SIZE));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_texture[2],
			      sizeof(uint32_t) * block_num * TBL_SIZE));
    r = num_data % num_unit;
    if (r != 0) {
	num_data = num_data + num_unit - r;
    }
    make_constant(MTGPDC_PARAM_TABLE, block_num);
    make_texture(MTGPDC_PARAM_TABLE, d_texture, block_num);
    make_kernel_data(d_status, MTGPDC_PARAM_TABLE, block_num);
    make_uint64_random(d_status, num_data, block_num);
    make_double_random(d_status, num_data, block_num);

    //finalize
    CUDA_SAFE_CALL(hipFree(d_status));
    CUDA_SAFE_CALL(hipFree(d_texture[0]));
    CUDA_SAFE_CALL(hipFree(d_texture[1]));
    CUDA_SAFE_CALL(hipFree(d_texture[2]));
#ifdef NEED_PROMPT
    CUT_EXIT(argc, argv);
#endif
}
