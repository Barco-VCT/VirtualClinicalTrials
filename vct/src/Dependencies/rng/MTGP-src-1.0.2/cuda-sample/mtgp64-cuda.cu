#include "hip/hip_runtime.h"
/**
 * @file mtgp64-cuda.cu
 *
 * @brief Sample Program for CUDA 2.2
 *
 * MTGP64-11213
 * This program generates 64-bit unsigned integers.
 * The period of generated integers is 2<sup>11213</sup>-1.
 *
 * This also generates double precision floating point numbers
 * uniformly distributed in the range [1, 2). (double r; 1.0 <= r < 2.0)
 *
 * @author Mutsuo Saito (Hiroshima University)
 * @author Makoto Matsumoto (Hiroshima University)
 *
 * Copyright (C) 2009 Mutsuo Saito, Makoto Matsumoto and
 * Hiroshima University. All rights reserved.
 *
 * The new BSD License is applied to this software, see LICENSE.txt
 */
#define __STDC_FORMAT_MACROS 1
#define __STDC_CONSTANT_MACROS 1
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <stdlib.h>
extern "C" {
#include "mtgp64-fast.h"
#include "mtgp64dc-param-11213.c"
}
#define MEXP 11213
#define N MTGPDC_N
#define THREAD_NUM MTGPDC_FLOOR_2P
#define LARGE_SIZE (THREAD_NUM * 3)
#define PARAM_NUM_MAX mtgpdc_params_11213_num
#define BLOCK_NUM_MAX 200
#define TBL_SIZE 16

/**
 * kernel I/O
 * This structure must be initialized before first use.
 */
struct mtgp64_kernel_status_t {
    uint64_t status[N];
};

/*
 * Generator Parameters.
 */
__constant__ uint32_t param_tbl[BLOCK_NUM_MAX][TBL_SIZE];
__constant__ uint32_t temper_tbl[BLOCK_NUM_MAX][TBL_SIZE];
__constant__ uint32_t double_temper_tbl[BLOCK_NUM_MAX][TBL_SIZE];
__constant__ uint32_t pos_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t sh1_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t sh2_tbl[BLOCK_NUM_MAX];
/* high_mask and low_mask should be set by make_constant(), but
 * did not work.
 */
__constant__ uint32_t high_mask = 0xffff8000;
__constant__ uint32_t low_mask =  0x00000000;

/**
 * Shared memory
 * The generator's internal status vector.
 */
__shared__ uint32_t status[2][LARGE_SIZE]; /* 512 * 3 elements, 12288 bytes. */

/**
 * The function of the recursion formula calculation.
 *
 * @param[out] RH 32-bit MSBs of output
 * @param[out] RL 32-bit LSBs of output
 * @param[in] X1H MSBs of the farthest part of state array.
 * @param[in] X1L LSBs of the farthest part of state array.
 * @param[in] X2H MSBs of the second farthest part of state array.
 * @param[in] X2L LSBs of the second farthest part of state array.
 * @param[in] YH MSBs of a part of state array.
 * @param[in] YL LSBs of a part of state array.
 * @param[in] bid block id.
 */
__device__ void para_rec(uint32_t *RH,
			 uint32_t *RL,
			 uint32_t X1H,
			 uint32_t X1L,
			 uint32_t X2H,
			 uint32_t X2L,
			 uint32_t YH,
			 uint32_t YL,
			 int bid) {
    uint32_t XH = (X1H & high_mask) ^ X2H;
    uint32_t XL = (X1L & low_mask) ^ X2L;
    uint32_t MAT;

    XH ^= XH << sh1_tbl[bid];
    XL ^= XL << sh1_tbl[bid];
    YH = XL ^ (YH >> sh2_tbl[bid]);
    YL = XH ^ (YL >> sh2_tbl[bid]);
    MAT = param_tbl[bid][YL & 0x0f];
    *RH = YH ^ MAT;
    *RL = YL;
}

/**
 * The tempering function.
 *
 * @param[in] VH MSBs of the output value should be tempered.
 * @param[in] VL LSBs of the output value should be tempered.
 * @param[in] TL LSBs of the tempering helper value.
 * @param[in] bid block id.
 * @return[in] the tempered value.
 */
__device__ uint64_t temper(uint32_t VH,
			   uint32_t VL,
			   uint32_t TL,
			   int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = temper_tbl[bid][TL & 0x0f];
    VH ^= MAT;
    r = ((uint64_t)VH << 32) | VL;
    return r;
}

/**
 * The tempering and converting function.
 * By using the preset-ted table, converting to IEEE format
 * and tempering are done simultaneously.
 *
 * @param[in] VH MSBs of the output value should be tempered.
 * @param[in] VL LSBs of the output value should be tempered.
 * @param[in] TL LSBs of the tempering helper value.
 * @param[in] bid block id.
 * @return the tempered and converted value.
 */
__device__ uint64_t temper_double(uint32_t VH,
				  uint32_t VL,
				  uint32_t TL,
				  int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = double_temper_tbl[bid][TL & 0x0f];
    r = ((uint64_t)VH << 32) | VL;
    r = (r >> 12) ^ ((uint64_t)MAT << 32);
    return r;
}

/**
 * Read the internal state vector from kernel I/O data, and
 * put them into shared memory.
 *
 * @param[out] status shared memory.
 * @param[in] d_status kernel I/O data
 * @param[in] bid block id
 * @param[in] tid thread id
 */
__device__ void status_read(uint32_t status[2][LARGE_SIZE],
			    const mtgp64_kernel_status_t *d_status,
			    int bid,
			    int tid) {
    uint64_t x;

    x = d_status[bid].status[tid];
    status[0][LARGE_SIZE - N + tid] = x >> 32;
    status[1][LARGE_SIZE - N + tid] = x & 0xffffffff;
    if (tid < N - THREAD_NUM) {
	x = d_status[bid].status[THREAD_NUM + tid];
	status[0][LARGE_SIZE - N + THREAD_NUM + tid] = x >> 32;
	status[1][LARGE_SIZE - N + THREAD_NUM + tid] = x & 0xffffffff;
    }
    __syncthreads();
}

/**
 * Read the internal state vector from shared memory, and
 * write them into kernel I/O data.
 *
 * @param[out] status shared memory.
 * @param[in] d_status kernel I/O data
 * @param[in] bid block id
 * @param[in] tid thread id
 */
__device__ void status_write(mtgp64_kernel_status_t *d_status,
			     const uint32_t status[2][LARGE_SIZE],
			     int bid,
			     int tid) {
    uint64_t x;

    x = (uint64_t)status[0][LARGE_SIZE - N + tid] << 32;
    x = x | status[1][LARGE_SIZE - N + tid];
    d_status[bid].status[tid] = x;
    if (tid < N - THREAD_NUM) {
	x = (uint64_t)status[0][4 * THREAD_NUM - N + tid] << 32;
	x = x | status[1][4 * THREAD_NUM - N + tid];
	d_status[bid].status[THREAD_NUM + tid] = x;
    }
    __syncthreads();
}

/**
 * kernel function.
 * This function generates 64-bit unsigned integers in d_data
 *
 * @param[in,out] d_status kernel I/O data
 * @param[out] d_data output
 * @param[in] size number of output data requested.
 */
__global__ void mtgp64_uint64_kernel(mtgp64_kernel_status_t* d_status,
				     uint64_t* d_data, int size) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t YH;
    uint32_t YL;
    uint64_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {

#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[0][LARGE_SIZE - N + tid]:%08x\n",
		   status[0][LARGE_SIZE - N + tid]);
	    printf("status[1][LARGE_SIZE - N + tid]:%08x\n",
		   status[1][LARGE_SIZE - N + tid]);
	    printf("status[0][LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[0][LARGE_SIZE - N + tid + 1]);
	    printf("status[1][LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[1][LARGE_SIZE - N + tid + 1]);
	    printf("status[0][LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[0][LARGE_SIZE - N + tid + pos]);
	    printf("status[1][LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[1][LARGE_SIZE - N + tid + pos]);
	    printf("sh1:%d\n", sh1_tbl[bid]);
	    printf("sh2:%d\n", sh2_tbl[bid]);
	    printf("high_mask:%08x\n", high_mask);
	    printf("low_mask:%08x\n", low_mask);
	    for (int j = 0; j < 16; j++) {
		printf("tbl[%d]:%08x\n", j, param_tbl[0][j]);
	    }
	}
#endif
	para_rec(&YH,
		 &YL,
		 status[0][LARGE_SIZE - N + tid],
		 status[1][LARGE_SIZE - N + tid],
		 status[0][LARGE_SIZE - N + tid + 1],
		 status[1][LARGE_SIZE - N + tid + 1],
		 status[0][LARGE_SIZE - N + tid + pos],
		 status[1][LARGE_SIZE - N + tid + pos],
		 bid);
	status[0][tid] = YH;
	status[1][tid] = YL;
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[0][tid]:%08x\n",	status[0][tid]);
	    printf("status[1][tid]:%08x\n",	status[1][tid]);
	}
#endif
	o = temper(YH,
		   YL,
		   status[1][LARGE_SIZE - N + tid + pos - 1],
		   bid);
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("o:%016" PRIx64 "\n", o);
	}
#endif
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + THREAD_NUM] = YH;
	status[1][tid + THREAD_NUM] = YL;
	o = temper(YH,
		   YL,
		   status[1][(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
		   bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][2 * THREAD_NUM - N + tid],
		 status[1][2 * THREAD_NUM - N + tid],
		 status[0][2 * THREAD_NUM - N + tid + 1],
		 status[1][2 * THREAD_NUM - N + tid + 1],
		 status[0][2 * THREAD_NUM - N + tid + pos],
		 status[1][2 * THREAD_NUM - N + tid + pos],
		 bid);
	status[0][tid + 2 * THREAD_NUM] = YH;
	status[1][tid + 2 * THREAD_NUM] = YL;
	o = temper(YH,
		   YL,
		   status[1][tid + pos - 1 + 2 * THREAD_NUM - N],
		   bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * kernel function.
 * This function generates double precision floating point numbers in d_data.
 *
 * @param[in,out] d_status kernel I/O data
 * @param[out] d_data output. IEEE double precision format.
 * @param[in] size number of output data requested.
 */
__global__ void mtgp64_double_kernel(mtgp64_kernel_status_t* d_status,
				     uint64_t* d_data, int size)
{

    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t YH;
    uint32_t YL;
    uint64_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {
	para_rec(&YH,
		 &YL,
		 status[0][LARGE_SIZE - N + tid],
		 status[1][LARGE_SIZE - N + tid],
		 status[0][LARGE_SIZE - N + tid + 1],
		 status[1][LARGE_SIZE - N + tid + 1],
		 status[0][LARGE_SIZE - N + tid + pos],
		 status[1][LARGE_SIZE - N + tid + pos],
		 bid);
	status[0][tid] = YH;
	status[1][tid] = YL;
	o = temper_double(YH,
			  YL,
			  status[1][LARGE_SIZE - N + tid + pos - 1],
			  bid);
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + THREAD_NUM] = YH;
	status[1][tid + THREAD_NUM] = YL;
	o = temper_double(
	    YH,
	    YL,
	    status[1][(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
	    bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][2 * THREAD_NUM - N + tid],
		 status[1][2 * THREAD_NUM - N + tid],
		 status[0][2 * THREAD_NUM - N + tid + 1],
		 status[1][2 * THREAD_NUM - N + tid + 1],
		 status[0][2 * THREAD_NUM - N + tid + pos],
		 status[1][2 * THREAD_NUM - N + tid + pos],
		 bid);
	status[0][tid + 2 * THREAD_NUM] = YH;
	status[1][tid + 2 * THREAD_NUM] = YL;
	o = temper_double(YH,
			  YL,
			  status[1][tid + pos - 1 + 2 * THREAD_NUM - N],
			  bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * This function sets constants in device memory.
 * @param[in] params input, MTGP64 parameters.
 */
void make_constant(const mtgp64_params_fast_t params[],
		   int block_num) {
    const int size1 = sizeof(uint32_t) * block_num;
    const int size2 = sizeof(uint32_t) * block_num * TBL_SIZE;
    uint32_t *h_pos_tbl;
    uint32_t *h_sh1_tbl;
    uint32_t *h_sh2_tbl;
    uint32_t *h_param_tbl;
    uint32_t *h_temper_tbl;
    uint32_t *h_double_temper_tbl;
#if 0
    uint32_t *h_high_mask;
    uint32_t *h_low_mask;
#endif
    h_pos_tbl = (uint32_t *)malloc(size1);
    h_sh1_tbl = (uint32_t *)malloc(size1);
    h_sh2_tbl = (uint32_t *)malloc(size1);
    h_param_tbl = (uint32_t *)malloc(size2);
    h_temper_tbl = (uint32_t *)malloc(size2);
    h_double_temper_tbl = (uint32_t *)malloc(size2);
#if 0
    h_high_mask = (uint32_t *)malloc(sizeof(uint32_t));
    h_low_mask = (uint32_t *)malloc(sizeof(uint32_t));
#endif
    if (h_pos_tbl == NULL
	|| h_sh1_tbl == NULL
	|| h_sh2_tbl == NULL
	|| h_param_tbl == NULL
	|| h_temper_tbl == NULL
	|| h_double_temper_tbl == NULL
#if 0
	|| h_high_mask == NULL
	|| h_low_mask == NULL
#endif
	) {
	printf("failure in allocating host memory for constant table.\n");
	exit(1);
    }
#if 0
    *h_high_mask = params[0].mask >> 32;
    *h_low_mask = params[0].mask & 0xffffffffU;
#endif
    for (int i = 0; i < block_num; i++) {
	h_pos_tbl[i] = params[i].pos;
	h_sh1_tbl[i] = params[i].sh1;
	h_sh2_tbl[i] = params[i].sh2;
	for (int j = 0; j < TBL_SIZE; j++) {
	    h_param_tbl[i * TBL_SIZE + j] = params[i].tbl[j] >> 32;
	    h_temper_tbl[i * TBL_SIZE + j] = params[i].tmp_tbl[j] >> 32;
	    h_double_temper_tbl[i * TBL_SIZE + j]
		= params[i].dbl_tmp_tbl[j] >> 32;
	}
    }
    // copy from malloc area only
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pos_tbl), h_pos_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sh1_tbl), h_sh1_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sh2_tbl), h_sh2_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(param_tbl), h_param_tbl, size2));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(temper_tbl), h_temper_tbl, size2));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(double_temper_tbl),
				      h_double_temper_tbl, size2));
#if 0
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&high_mask),
				      &h_high_mask, sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&low_mask),
				      &h_low_mask, sizeof(uint32_t)));
#endif
    free(h_pos_tbl);
    free(h_sh1_tbl);
    free(h_sh2_tbl);
    free(h_param_tbl);
    free(h_temper_tbl);
    free(h_double_temper_tbl);
#if 0
    free(h_high_mask);
    free(h_low_mask);
#endif
}

#include "mtgp-cuda-common.c"
#include "mtgp64-cuda-common.c"

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
void make_uint64_random(mtgp64_kernel_status_t* d_status,
			int num_data,
			int block_num) {
    uint64_t* d_data;
    unsigned int timer = 0;
    uint64_t* h_data;
    hipError_t e;
    float gputime;

    printf("generating 64-bit unsigned random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(uint64_t) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (uint64_t *) malloc(sizeof(uint64_t) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp64_uint64_kernel<<< block_num, THREAD_NUM>>>(
	d_status, d_data, num_data / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint64_t) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_uint64_array(h_data, num_data, block_num);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
void make_double_random(mtgp64_kernel_status_t* d_status,
			int num_data,
			int block_num) {
    uint64_t* d_data;
    unsigned int timer = 0;
    double* h_data;
    hipError_t e;
    float gputime;

    printf("generating double precision floating point random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(uint64_t) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (double *) malloc(sizeof(double) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp64_double_kernel<<< block_num, THREAD_NUM >>>(
	d_status, d_data, num_data / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint64_t) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_double_array(h_data, num_data, block_num);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

int main(int argc, char** argv)
{
    // LARGE_SIZE is a multiple of 16
    int num_data = 10000000;
    int block_num;
    int block_num_max;
    int num_unit;
    int r;
    mtgp64_kernel_status_t* d_status;

    if (argc >= 2) {
	errno = 0;
	block_num = strtol(argv[1], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	if (BLOCK_NUM_MAX < PARAM_NUM_MAX) {
	    block_num_max = BLOCK_NUM_MAX;
	} else {
	    block_num_max = PARAM_NUM_MAX;
	}
	if (block_num < 1 || block_num > block_num_max) {
	    printf("%s block_num should be between 1 and %d\n",
		   argv[0], block_num_max);
	    return 1;
	}
	errno = 0;
	num_data = strtol(argv[2], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	argc -= 2;
	argv += 2;
    } else {
	CUT_DEVICE_INIT(argc, argv);
	printf("%s number_of_block number_of_output\n", argv[0]);
	block_num = get_suitable_block_num(sizeof(uint64_t),
					   THREAD_NUM,
					   LARGE_SIZE);
	if (block_num <= 0) {
	    printf("can't calculate sutable number of blocks.\n");
	    return 1;
	}
	printf("the suitable number of blocks for device 0 "
	       "will be multiple of %d\n", block_num);
	return 1;
    }
    CUT_DEVICE_INIT(argc, argv);
    num_unit = LARGE_SIZE * block_num;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_status,
			      sizeof(mtgp64_kernel_status_t) * block_num));
    r = num_data % num_unit;
    if (r != 0) {
	num_data = num_data + num_unit - r;
    }
    make_constant(MTGPDC_PARAM_TABLE, block_num);
    make_kernel_data(d_status, MTGPDC_PARAM_TABLE, block_num);
    make_uint64_random(d_status, num_data, block_num);
    make_double_random(d_status, num_data, block_num);

    //finalize
    CUDA_SAFE_CALL(hipFree(d_status));
#ifdef NEED_PROMPT
    CUT_EXIT(argc, argv);
#endif
}
